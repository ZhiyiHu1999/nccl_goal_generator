#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

static uint64_t getHostHash(const char* string) {
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}

static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage: %s <size>\n", argv[0]);
    return -1;
  }

  int size = atoi(argv[1]); 

  int myRank, nRanks, localRank = 0;

  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }

  printf("The local rank is: %d\n", localRank);

  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;

  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
  
  CUDACHECK(hipMemset(sendbuff, 0, size * sizeof(float)));
  CUDACHECK(hipMemset(recvbuff, 0, size * sizeof(float)));
 
  CUDACHECK(hipStreamCreate(&s));

  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  NCCLCHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclSum, comm, s));

  CUDACHECK(hipStreamSynchronize(s));

  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));

  ncclCommDestroy(comm);

  MPICHECK(MPI_Finalize());

  printf("[MPI Rank %d] Success \n", myRank);
  
  return 0;
}
